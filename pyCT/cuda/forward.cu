#include "hip/hip_runtime.h"
#include "forward.h"

__global__ 
void kernel_parallel(float* proj, int nw, hipTextureObject_t texObjImg, float* transformation)
{
	int nu = gridDim.x;
	int nv = gridDim.y;
	int iu = blockIdx.x;
	int iv = blockIdx.y;
	
	// int na = blockDim.x;
	int ia = threadIdx.x;
	
	float u = -1. + (float) (1 + iu * 2) / nu;
	float v = -1. + (float) (1 + iv * 2) / nv;
	float w = -1. + (float) 1/nw;
	float dw = (float) 2/nw;

	float t00 = transformation[0 + 0*4 + ia*4*4];
	float t01 = transformation[1 + 0*4 + ia*4*4];
	float t02 = transformation[2 + 0*4 + ia*4*4];
	float t03 = transformation[3 + 0*4 + ia*4*4];

	float t10 = transformation[0 + 1*4 + ia*4*4];
	float t11 = transformation[1 + 1*4 + ia*4*4];
	float t12 = transformation[2 + 1*4 + ia*4*4];
	float t13 = transformation[3 + 1*4 + ia*4*4];

	float t20 = transformation[0 + 2*4 + ia*4*4];
	float t21 = transformation[1 + 2*4 + ia*4*4];
	float t22 = transformation[2 + 2*4 + ia*4*4];
	float t23 = transformation[3 + 2*4 + ia*4*4];
	
	float xx = t00 * u + t01 * v + t03;
	float yy = t10 * u + t11 * v + t13;
	float zz = t20 * u + t21 * v + t23;

	float sum = 0;

	for (int i = 0; i < nw; i++)
	{
		float x = xx + t02 * w;
		float y = yy + t12 * w;
		float z = zz + t22 * w;
		sum += tex3D<float>(texObjImg, x+.5, y+.5, z+.5);
		w += dw;
	}
	int idx = iu + iv*nu + ia*nu*nv;
	proj[idx] = sum;
}

void parallel_cuda(float *detector_array, float *transformation, float *object_array, int nx, int ny, int nz, int nu, int nv, int nw, int na)
{
	// object array >> texture memory
    const hipExtent objSize = make_hipExtent(nz, ny, nx);
    hipArray *d_object_array = 0;
    hipTextureObject_t tex_object_array = 0;

	// create 3D array
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipMalloc3DArray(&d_object_array, &channelDesc, objSize);
    
	// copy data to 3D array
	hipMemcpy3DParms copyParams = { 0 };
	copyParams.srcPtr = make_hipPitchedPtr((void*)object_array, objSize.width * sizeof(float), objSize.width, objSize.height);
	copyParams.dstArray = d_object_array;
	copyParams.extent = objSize;
	copyParams.kind = hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParams);

	hipResourceDesc            texRes;
	memset(&texRes, 0, sizeof(hipResourceDesc));
	texRes.resType = hipResourceTypeArray;
	texRes.res.array.array = d_object_array;

	hipTextureDesc             texDescr;
	memset(&texDescr, 0, sizeof(hipTextureDesc));
	texDescr.normalizedCoords = false; // access with normalized texture coordinates
	texDescr.filterMode = hipFilterModeLinear; // linear interpolation
	texDescr.addressMode[0] = hipAddressModeBorder; // wrap texture coordinates
	texDescr.addressMode[1] = hipAddressModeBorder; // wrap texture coordinates
	texDescr.addressMode[2] = hipAddressModeBorder; // wrap texture coordinates
	texDescr.readMode = hipReadModeElementType;

	hipCreateTextureObject(&tex_object_array, &texRes, &texDescr, NULL);

	//
	float *d_transformation;
	hipMalloc(&d_transformation, na * 4 * 4 * sizeof(float));
	hipMemcpy(d_transformation, transformation, na * 4 * 4 * sizeof(float), hipMemcpyHostToDevice);
	//
	float *d_detector_array;
    hipMalloc(&d_detector_array, na * nu * nv * sizeof(float));
	//
	kernel_parallel <<< dim3(nu,nv,1), dim3(na,1,1) >>> (d_detector_array, nw, tex_object_array, d_transformation);
    hipMemcpy(detector_array, d_detector_array, na*nu*nv*sizeof(float), hipMemcpyDeviceToHost);
    
	
	hipFree(d_detector_array);
	hipFree(d_transformation);
	hipFreeArray(d_object_array);
	hipDestroyTextureObject(tex_object_array);
}